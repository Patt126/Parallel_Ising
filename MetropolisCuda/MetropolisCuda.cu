#include "hip/hip_runtime.h"
/*The structure of this CUDA program is specifically designed for development on cloud-based platforms like Google Colab.
The streamlined code structure, avoiding traditional `.cuh` and `.cu` files and `CMake`, is more practical for these environments,
 focusing on ease of use and efficiency in building and executing the program.
*/

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>

/*

This CUDA program is designed with flexibility in mind, allowing for key parameters to be adjusted according to the specific requirements of the simulation.
 The primary configurable parameters include:

L (Lattice Size): Determines the dimensions of the lattice (N = L * L). It's important that L is a multiple of two for proper lattice structure.

J (Interaction Strength): This parameter influences the interaction strength within the system, and can be modified to study various physical behaviors.

NTHREADS (Number of GPU Threads): Critical for performance optimization. Choose a value that allows the GPU to efficiently manage computation.

IT (Number of Iterations): Defines the total number of iterations for the simulation, crucial for ensuring the accuracy and convergence of the results.

Performance and Convergence Guidelines

Thread Count and Lattice Size: It's recommended that L be divisible by NTHREADS to ensure efficient workload distribution and to avoid indexing issues.
If you modify L, adjust NTHREADS accordingly to maintain this divisibility.

Optimal Settings: The program was tested with L = 256, IT = 2e9, and NTHREADS = 256. These settings are known to provide a high confidence of convergence in simulations.
Adjusting for Larger Lattices: If you choose to increase L, be mindful that it might require a proportional adjustment in NTHREADS and IT depending on your GPU capabilities.
A larger L generally necessitates more iterations to ensure convergence, and the thread count may need to be modified to maintain optimal performance.



*/

#define L 256
#define N (L*L)
#define J 1.00
#define IT 2e9 // Number of iterations, should be divisible by 2 for even updates
#define NTHREADS 128 // Number of GPU threads

__device__ int get_index(int row, int col);
__device__ int delta_energy(bool* lattice, int r, int c);
__global__ void flip_spins(bool* lattice, float* prob, float* energy, hiprandState* states, bool update_black);
__global__ void setup_rand_kernel(hiprandState* state, unsigned long seed);
__global__ void initialize_lattice_kernel(bool* lattice, hiprandState* states);
__global__ void calculate_magnetization_kernel(bool* lattice, float* magnetization);


int main() {
    bool* dev_lattice;
    hipMalloc((void**)&dev_lattice, N * sizeof(bool));

    float* dev_energy;
    hipMalloc((void**)&dev_energy, sizeof(float));

    float* dev_magnetization;
    hipMalloc((void**)&dev_magnetization, sizeof(float));

    hiprandState* dev_states;
    hipMalloc((void**)&dev_states, N * sizeof(hiprandState));

    dim3 blocksPerGrid((N + NTHREADS - 1) / NTHREADS, 1, 1);
    dim3 threadsPerBlock(NTHREADS, 1, 1);

    unsigned long seed = static_cast<unsigned long>(time(nullptr));
    setup_rand_kernel << < blocksPerGrid, threadsPerBlock >> > (dev_states, seed);

    initialize_lattice_kernel << < blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_states);

    float* dev_probabilities;
    hipMalloc((void**)&dev_probabilities, 2 * sizeof(float));

    for (float T = 0.2f; T <= 3.0f; T += 0.1f) {
        clock_t start_time = clock();

        float energy = 0.0f;
        hipMemcpy(dev_energy, &energy, sizeof(float), hipMemcpyHostToDevice);

        float magnetization = 0.0f;
        hipMemcpy(dev_magnetization, &magnetization, sizeof(float), hipMemcpyHostToDevice);

        float prob[2] = { exp(-4 * J / T), exp(-8 * J / T) };
        hipMemcpy(dev_probabilities, prob, 2 * sizeof(float), hipMemcpyHostToDevice);

        for (unsigned long i = 0; i < IT / N; i += 2) {
            flip_spins << < blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_probabilities, dev_energy, dev_states, true);
            flip_spins << < blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_probabilities, dev_energy, dev_states, false);
        }

        calculate_magnetization_kernel << < blocksPerGrid, threadsPerBlock >> > (dev_lattice, dev_magnetization);
        hipDeviceSynchronize();

        hipMemcpy(&energy, dev_energy, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&magnetization, dev_magnetization, sizeof(float), hipMemcpyDeviceToHost);
        magnetization /= N;

        clock_t end_time = clock();
        double elapsed_secs = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

        std::cout << "Temperature: " << T << std::endl;
        std::cout << "Magnetization per site: " << abs(magnetization) << std::endl;
        std::cout << "Simulation time (seconds): " << elapsed_secs << std::endl << std::endl;
    }

    hipFree(dev_lattice);
    hipFree(dev_energy);
    hipFree(dev_magnetization);
    hipFree(dev_states);
    hipFree(dev_probabilities);

    return 0;
}



__device__ int get_index(int row, int col) {
    return (row * L + col) % N;
}

// The lattice uses boolean values, true for spin up (equivalent to 1) and false for spin down (equivalent to -1)
__device__ int delta_energy(bool* lattice, int r, int c) {
    int sum = lattice[get_index((r - 1 + L) % L, c)]
        + lattice[get_index((r + 1) % L, c)]
        + lattice[get_index(r, (c - 1 + L) % L)]
        + lattice[get_index(r, (c + 1) % L)];
    sum = 2 * sum - 4; // Convert sum from [0, 4] to [-4, 4] to match the original spin values
    int spin = lattice[get_index(r, c)] ? 1 : -1; // Convert bool to equivalent spin value
    return 2 * spin * sum;
}

__global__ void flip_spins(bool* lattice, float* prob, float* energy, hiprandState* states, bool update_black) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    int r = idx / L;
    int c = idx % L;
    bool is_black = ((r + c) % 2 == 0);

    if (is_black == update_black) {
        int delta = delta_energy(lattice, r, c);
        float rnd = hiprand_uniform(&states[idx]);

        if (delta <= 0 || (delta == 4 && rnd < prob[0]) || (delta == 8 && rnd < prob[1])) {
            lattice[get_index(r, c)] = !lattice[get_index(r, c)];
            atomicAdd(energy, delta * J);
            // Removed magnetization update
        }
    }
}

__global__ void initialize_lattice_kernel(bool* lattice, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float randVal = hiprand_uniform(&states[idx]);
        lattice[idx] = (randVal < 0.5f);
        // Removed magnetization calculation and update
    }
}

__global__ void setup_rand_kernel(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

__global__ void calculate_magnetization_kernel(bool* lattice, float* magnetization) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int spin = lattice[idx] ? 1 : -1; // Convert boolean to +1 or -1
        atomicAdd(magnetization, spin);   // Add the spin to the total magnetization
    }
}
